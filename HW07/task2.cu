#include "hip/hip_runtime.h"
// task2.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "count.cuh"


int main(int argc, char *argv[]) {

    // Create the variables to receive the input number
    int n = static_cast<int>(pow(2, atoi(argv[1])));

    // Create a host vector of integers with size 'n'
    thrust::host_vector<int> hdata(n);

    // Create a default random number generator
    thrust::default_random_engine rng;

    // Create a uniform integer distribution that generates random numbers
    // in the range [0, 500]
    thrust::uniform_int_distribution<int> dist(0, 500);

    // Fill the host vector with random integer values
    for (int i = 0; i < n; i++) {

        // Generate a random integer 
        // using the 'dist' distribution and 'rng' generator
        hdata[i] = dist(rng);
    }

    // Create a device vector and copy the data from the host vector
    thrust::device_vector<int> ddata = hdata;

    // Create device vectors 'values' and 'counts' to store results
    thrust::device_vector<int> values;
    thrust::device_vector<int> counts;

    // Measure time with CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start the timer
    hipEventRecord(start);

    // Call the count function
    count(ddata, values, counts);

    // Stop th timer
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Print the last element of values and counts
    std::cout << values[values.size() - 1] << std::endl;
    std::cout << counts[counts.size() - 1] << std::endl;

    // Calculate duration time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    
    // Print the time taken to run the count function in milliseconds
    std::cout << milliseconds << std::endl;

    return 0;

}






