// count.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include "count.cuh"


// Find unique integers in the array d_in, store them in values array in ascending order,
// and store the occurrences of these integers in counts array.
void count(const thrust::device_vector<int>& d_in, thrust::device_vector<int>& values, thrust::device_vector<int>& counts) {
    
    // Make a copy of the input vector and sort it
    thrust::device_vector<int> sorted_in = d_in;
    thrust::sort(sorted_in.begin(), sorted_in.end());

    // Find unique elements in the sorted vector
    thrust::device_vector<int>::iterator new_end = thrust::unique(sorted_in.begin(), sorted_in.end());

    // Resize the 'values' and 'counts' vectors to accommodate the unique elements
    int unique_count = new_end - sorted_in.begin();
    values.resize(unique_count);
    counts.resize(unique_count);

    // Compute the counts using the 'reduce by key' operation
    thrust::reduce_by_key(sorted_in.begin(), new_end, thrust::constant_iterator<int>(1), values.begin(), counts.begin());
    
}






