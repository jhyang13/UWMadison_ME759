#include "hip/hip_runtime.h"
// task1_thrust.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>


int main(int argc, char **argv) {

	// Create the variables to receive the input number
	// n is used to decide the size of matrix
	int n = pow(2, atoi(argv[1]));

	// Create a host vector to store random float numbers
    	thrust::host_vector<float> hdata(n);
	// Create a random number generator engine
    	thrust::default_random_engine rng;
	// Define a uniform distribution for random numbers in the range [-1.0, 1.0]
    	thrust::uniform_real_distribution<float> dist(-1.0f, 1.0f);
	// Fill the host vector with random numbers using the random number generator and distribution
    	thrust::generate(hdata.begin(), hdata.end(), [&]() { return dist(rng); });

	// Copy the host vector to a device vector
    	thrust::device_vector<float> ddata = hdata;

    	// Set up timer
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

	// Start timer
    	hipEventRecord(start);

    	// Perform reduction
    	float result = thrust::reduce(ddata.begin(), ddata.end(), 0.0f, thrust::plus<float>());

	// Stop timer
    	hipEventRecord(stop);
    	hipEventSynchronize(stop);

	// Create a variable to store the duration time in milliseconds
    	float milliseconds = 0.0f;
    	hipEventElapsedTime(&milliseconds, start, stop);

    	// Print the result 
    	std::cout << result << std::endl;
	// Print the time
    	std::cout << milliseconds << std::endl;

    	return 0;
}




