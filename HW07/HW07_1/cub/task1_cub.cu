#include "hip/hip_runtime.h"
// task1_cub.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

// Print CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <thrust/reduce.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>
#include <stdio.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_reduce.cuh>
#include "cub/util_debug.cuh"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

// Import the CUB library's namespace to access its elements
using namespace hipcub;

// Define a caching allocator for device memory
CachingDeviceAllocator  g_allocator(true);

int main(int argc, char** argv) {
    
    // Create the variables to receive the input number
    // n is used to decide the size of matrix
    int n = pow(2, atoi(argv[1]));

    // Create and populate a host vector with random numbers
    // Create a host vector of floats with size 'n'
    thrust::host_vector<float> h_in(n);
    // Create a random number generator engine
    thrust::default_random_engine rng;
    // Define a uniform distribution between -1.0 and 1.0
    thrust::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    // Fill the vector with random numbers within the defined range
    thrust::generate(h_in.begin(), h_in.end(), [&]() { return dist(rng); });

    // Set up device arrays
    float* d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_in, sizeof(float) * n));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in.data(), sizeof(float) * n, hipMemcpyHostToDevice));
    
    // Setup device output array
    float* d_sum = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)& d_sum, sizeof(float) * 1));

    // Request and allocate temporary storage
    // Pointer to temporary storage
    void* d_temp_storage = NULL;
    // Variable to hold the size of temporary storage needed
    size_t temp_storage_bytes = 0;

    // Determine the required temporary storage size for the reduction
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));

    // Allocate device memory for the determined temporary storage
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Start recording
    hipEventRecord(start);

    // Do the actual reduce operation
    CubDebugExit(DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, d_in, d_sum, n));
    float gpu_sum;
    CubDebugExit(hipMemcpy(&gpu_sum, d_sum, sizeof(float) * 1, hipMemcpyDeviceToHost));

    // Stop recording
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Check for correctness
    printf("%f\n", gpu_sum);
    // Print the time taken in milliseconds
    printf("%.3f\n", milliseconds);

    // Cleanup
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_sum) CubDebugExit(g_allocator.DeviceFree(d_sum));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    
    return 0;

}




