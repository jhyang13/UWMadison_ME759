#include "hip/hip_runtime.h"
// task2.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "stencil.cuh"


int main(int argc, char **argv) {

	// Create the variables to receive the input number
	unsigned int N = pow(2, atoi(argv[1]));
	unsigned int R = atoi(argv[2]);
	unsigned int Threads_per_Block = atoi(argv[3]);
	
 	// Assign host memory and stack allocation for array image, mask and output
	float* hi = new float[N];
	float* hm = new float[2 * R + 1];
	float* hO = new float[N];

	// Create Pseudo-random number generator 
	// used for generating pseudo-random number sequences
        std::default_random_engine generator(std::time(nullptr));
	// Set up random numbers in the range [-1, 1]
	std::uniform_real_distribution<float> distribution(-1.0f, 1.0f);

	// Assign values to array image with the created random values
    	for (unsigned int i = 0; i < N; ++i){

        	hi[i] = distribution(generator);

    	}

	// Assign values to array mask with the created random values
	for (unsigned int j = 0; j < 2 * R + 1; ++j){

        	hm[j] = distribution(generator);

    	}

    	// Allocate device memory for array a, b and c
    	float *di; 
	float *dm;
	float *dO;

    	hipMalloc((void**)&di, sizeof(float) * N);
	hipMalloc((void**)&dm, sizeof(float) * (2 * R + 1));
	hipMalloc((void**)&dO, sizeof(float) * N);


	/*
	// Test if hipMalloc is incorrect
	hipError_t cudaStatus = hipMalloc((void**)&di, sizeof(float) * N);
	if (cudaStatus != hipSuccess) {
    		std::cerr << "hipMalloc failed: " << hipGetErrorString(cudaStatus) << std::endl;
	}*/


    	// Copy back the data stored in the device array
        // into the host array for array a and b
    	hipMemcpy(di, hi, sizeof(float) * N, hipMemcpyHostToDevice);
    	hipMemcpy(dm, hm, sizeof(float) * (2 * R + 1), hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Lanuch the timer
	hipEventRecord(start);

   	// Call the matmul function
    	stencil(di, dm, dO, N, R, Threads_per_Block);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
        hipDeviceSynchronize();

    	// Copy back the data stored in the device array
        // into the host array for array c
    	hipMemcpy(hO, dO, sizeof(float) * N, hipMemcpyDeviceToHost);


	/*
	// Test if data can be transferred
	hipMemcpy(hi, di, sizeof(float) * N, hipMemcpyDeviceToHost);

	// Just output several data
	for (int i = 0; i < 10; ++i) {
    		printf("hi[%d] = %f\n", i, hi[i]);
	}*/


	// Stop the timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration time in milliseconds
    	float milliseconds = 0.0f;
    	hipEventElapsedTime(&milliseconds, start, stop);


	// Print the last element of the resulting array
	std::cout << hO[N - 1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
        std::cout << milliseconds << std::endl;
	
	// Free memory
    	delete[] hi;
    	delete[] hm;
    	delete[] hO;
    	hipFree(di);
    	hipFree(dm);
    	hipFree(dO);

    	return 0;

}





