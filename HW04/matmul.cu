#include "hip/hip_runtime.h"
// matmul.cu
        
/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/
        
#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h> 
#include "matmul.cuh"


__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n){

	// Calculates the row and column coordinates of the element 
	// that the current thread is processing in the result matrix
	int row = blockIdx.x * blockDim.x + threadIdx.x;
    	int col = blockIdx.y * blockDim.y + threadIdx.y;

	// Check if the thread is within valid row and column values
	if (row < n && col < n) {

		// Create a variable to store the value
        	float result = 0.0f;
        
		// Loop to assign value to array C
		for (int i = 0; i < n; ++i){
            		
			result += A[row * n + i] * B[i * n + col];

        	}
        
		// Assign value to array C
		C[row * n + col] = result;

	}
}
    	

void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block){

	// Define a 2D block
	dim3 block(threads_per_block, threads_per_block);

	// Define a 2D grid
    	dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);

	// Call the kernel function 
    	matmul_kernel<<<grid, block>>>(A, B, C, n);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
    	hipDeviceSynchronize();
}





