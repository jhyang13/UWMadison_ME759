#include "hip/hip_runtime.h"
// stencil.cuh

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/
        
#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h> 
#include "stencil.cuh"


// Without using shared memory
/*
__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    
	// Calculate the element in the result matrix 
	// that the current thread is processing
    	int i = blockIdx.x * blockDim.x + threadIdx.x;

    	// Check if the thread is within that block
    	if (i < n) {

        	// Create a variable to save the result
        	float result = 0.0f;

        	// Loop to calculate the convolution
        	for (int j = -R; j <= R; ++j) {

            		if (i + j >= 0 && i + j < n) {

                		result += image[i + j] * mask[j + R];

            		} else {

                		// When i + j < 0 or i + j > n - 1, image[i] = 1
                		result += 1.0f * mask[j + R];
            		}
        	}

        	// Assign value to output
        	output[i] = result;
    	}
}
*/


__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    
    	// Shared memory declaration
    	__shared__ float shared_image[1024]; // THREADS_PER_BLOCK = 1024
    
    	// Calculate the element in the result matrix 
	// that the current thread is processing
    	int i = blockIdx.x * blockDim.x + threadIdx.x;

    	// Add image to the shared memory
    	if (threadIdx.x < n) {
        	shared_image[threadIdx.x] = image[i];
    	}

    	// Wait for all threads to complete shared memory loading
    	__syncthreads();

    	// Check if the thread is within a valid value range
    	if (i < n) {

        	// Create a variable to save result
        	float result = 0.0f;

        	// Loop to get the convolution
        	for (int j = -R; j <= R; ++j) {

            		int idx = threadIdx.x + j;

            		if (idx >= 0 && idx < n) {

                		result += shared_image[idx] * mask[j + R];

            		} else {
                	
                		result += 1.0f * mask[j + R];

            		}
        	}

        // Assign value to output
        output[i] = result;
    }
}



__host__ void stencil(const float* image, const float* mask, float* output, unsigned int n, unsigned int R, unsigned int threads_per_block){

	// Define grid and block dimensions	
	// Define a 2D block
        dim3 block(threads_per_block);

        // Define a 2D grid
        dim3 grid((n + block.x - 1) / block.x);

        // Call the kernel function
        stencil_kernel<<<grid, block>>>(image, mask, output, n, R);

        // Tell the host waits for the kernel
        // to finish printing before returning from main
       	hipDeviceSynchronize();

}

