#include "hip/hip_runtime.h"
// task1.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "matmul.cuh"


int main(int argc, char **argv) {

	// Create the variables to receive the input number
	unsigned int N = pow(2, atoi(argv[1]));
	unsigned int Threads_per_Block = atoi(argv[2]);
	
 	// Assign host memory and stack allocation for array a, b and c
	float* ha = new float[N * N];
	float* hb = new float[N * N];
	float* hc = new float[N * N];

	// Create Pseudo-random number generator 
	// used for generating pseudo-random number sequences
        std::default_random_engine generator(std::time(nullptr));
	// Set up random numbers in the range [-1, 1]
	std::uniform_real_distribution<float> distribution(-1.0f, 1.0f);

	// Assign values to array a and b with the created random values
    	for (unsigned int i = 0; i < N * N; ++i){
        	ha[i] = distribution(generator);
        	hb[i] = distribution(generator);
    	}

    	// Allocate device memory for array a, b and c
    	float *da; 
	float *db;
	float *dc;

    	hipMalloc((void**)&da, sizeof(float) * N * N);
	hipMalloc((void**)&db, sizeof(float) * N * N);
	hipMalloc((void**)&dc, sizeof(float) * N * N);

    	// Copy back the data stored in the device array
        // into the host array for array a and b
    	hipMemcpy(da, ha, sizeof(float) * N * N, hipMemcpyHostToDevice);
    	hipMemcpy(db, hb, sizeof(float) * N * N, hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Lanuch the timer
	hipEventRecord(start);

   	// Call the matmul function
    	matmul(da, db, dc, N, Threads_per_Block);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
        hipDeviceSynchronize();

    	// Copy back the data stored in the device array
        // into the host array for array c
    	hipMemcpy(hc, dc, sizeof(float) * N * N, hipMemcpyDeviceToHost);


	// Check if the kernel function call is in error
	/*hipError_t error = hipGetLastError();
	//if (error != hipSuccess) {
    		std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
	}*/


	// Stop the timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration time in milliseconds
    	float milliseconds = 0.0f;
    	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the last element of the resulting array
	std::cout << hc[N * N - 1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
        std::cout << milliseconds << std::endl;
	
	// Free memory
    	delete[] ha;
    	delete[] hb;
    	delete[] hc;
    	hipFree(da);
    	hipFree(db);
    	hipFree(dc);

    	return 0;

}





