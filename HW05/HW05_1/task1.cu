#include "hip/hip_runtime.h"
// task1.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "matmul.cuh"


int main(int argc, char **argv) {

	// Create the variables to receive the input number
	// 2^n is used to decide the size of matrix
	// block_dim^2 should be the number of threads
	unsigned int n = pow(2, atoi(argv[1]));
	unsigned int block_dim  = atoi(argv[2]);



	// int part
    	// Allocate memory for matrices A, B, and C
	// Create new matrices A, B and C
    	int* ha = new int[n * n];
    	int* hb = new int[n * n];
    	int* hc = new int[n * n];

    	// Initialize the pseudo-random number generator's seed 
	// for generating random numbers in the program
    	srand(static_cast<unsigned int>(time(nullptr)));
	// Initialize matrices A and B with random values
	// From 0 to 50
    	for (unsigned int i = 0; i < n * n; ++i) {

        	ha[i] = static_cast<int>(rand() % 50); 
        	hb[i] = static_cast<int>(rand() % 50);

    	}

    	// Create device matrices A, B, and C
    	int* da;
    	int* db;
    	int* dc;
	// Allocate device memory for matrices A, B, and C
    	hipMalloc((void**)&da, sizeof(int) * n * n);
    	hipMalloc((void**)&db, sizeof(int) * n * n);
    	hipMalloc((void**)&dc, sizeof(int) * n * n);

    	// Copy matrices A and B to device
    	hipMemcpy(da, ha, sizeof(int) * n * n, hipMemcpyHostToDevice);
    	hipMemcpy(db, hb, sizeof(int) * n * n, hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Lanuch the timer
	hipEventRecord(start);

    	// Call the appropriate matmul function based on the template type
    	matmul_1(da, db, dc, n, block_dim);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
    	hipDeviceSynchronize();

    	// Copy back the data stored in the device array
    	// into the host array for array c
    	hipMemcpy(hc, dc, sizeof(int) * n * n, hipMemcpyDeviceToHost);

	// Stop the timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration time in milliseconds
    	float milliseconds = 0.0f;
    	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the first element of C
 	std::cout << hc[0] << std::endl;

	// Print the last element of C
	std::cout << hc[n * n - 1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
    	std::cout << milliseconds << std::endl;

    	// Free memory
    	delete[] ha;
    	delete[] hb;
    	delete[] hc;
    	hipFree(da);
    	hipFree(db);
    	hipFree(dc);



	// float part
    	// Allocate memory for matrices A, B, and C
	// Create new matrices A, B and C
    	float* ha2 = new float[n * n];
    	float* hb2 = new float[n * n];
    	float* hc2 = new float[n * n];

    	// Initialize the pseudo-random number generator's seed 
	// for generating random numbers in the program
    	srand(static_cast<unsigned int>(time(nullptr)));
	// Initialize matrices A and B with random values
	// From 0 to 50
    	for (unsigned int i = 0; i < n * n; ++i) {
        	
		// Integer part
		int integerPart = rand() % 50;    
		// Decimal part
    		float decimalPart = static_cast<float>(rand() % 100) / 100.0f;
		// Assign elements to ha2 and hb2
    		ha2[i] = static_cast<float>(integerPart) + decimalPart;
    		hb2[i] = static_cast<float>(integerPart) + decimalPart;

    	}

    	// Create device matrices A, B, and C
    	float* da2;
    	float* db2;
    	float* dc2;
	// Allocate device memory for matrices A, B, and C
    	hipMalloc((void**)&da2, sizeof(float) * n * n);
    	hipMalloc((void**)&db2, sizeof(float) * n * n);
    	hipMalloc((void**)&dc2, sizeof(float) * n * n);

    	// Copy matrices A and B to device
    	hipMemcpy(da2, ha2, sizeof(float) * n * n, hipMemcpyHostToDevice);
    	hipMemcpy(db2, hb2, sizeof(float) * n * n, hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start2;
	hipEvent_t stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	// Lanuch the timer
	hipEventRecord(start2);

    	// Call the appropriate matmul function based on the template type
    	matmul_2(da2, db2, dc2, n, block_dim);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
    	hipDeviceSynchronize();

    	// Copy back the data stored in the device array
    	// into the host array for array c
    	hipMemcpy(hc2, dc2, sizeof(float) * n * n, hipMemcpyDeviceToHost);

	// Stop the timer
	hipEventRecord(stop2);
	hipEventSynchronize(stop2);

	// Calculate the duration time in milliseconds
    	float milliseconds2 = 0.0f;
    	hipEventElapsedTime(&milliseconds2, start2, stop2);

	// Print the first element of C
 	std::cout << hc2[0] << std::endl;

	// Print the last element of C
	std::cout << hc2[n * n - 1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
    	std::cout << milliseconds2 << std::endl;

    	// Free memory
    	delete[] ha2;
    	delete[] hb2;
    	delete[] hc2;
    	hipFree(da2);
    	hipFree(db2);
    	hipFree(dc2);


		
	// double part
    	// Allocate memory for matrices A, B, and C
	// Create new matrices A, B and C
    	double* ha3 = new double[n * n];
    	double* hb3 = new double[n * n];
    	double* hc3 = new double[n * n];

    	// Initialize the pseudo-random number generator's seed 
	// for generating random numbers in the program
    	srand(static_cast<unsigned int>(time(nullptr)));
	// Initialize matrices A and B with random values
	// From 0 to 50
    	for (unsigned int i = 0; i < n * n; ++i) {
        	
		// Integer part
		int integerPart = rand() % 50;    
		// Decimal part
    		double decimalPart = static_cast<double>(rand() % 100) / 100.0f;
		// Assign elements to ha2 and hb2
    		ha3[i] = static_cast<double>(integerPart) + decimalPart;
    		hb3[i] = static_cast<double>(integerPart) + decimalPart;

    	}

    	// Create device matrices A, B, and C
    	double* da3;
    	double* db3;
    	double* dc3;
	// Allocate device memory for matrices A, B, and C
    	hipMalloc((void**)&da3, sizeof(double) * n * n);
    	hipMalloc((void**)&db3, sizeof(double) * n * n);
    	hipMalloc((void**)&dc3, sizeof(double) * n * n);

    	// Copy matrices A and B to device
    	hipMemcpy(da3, ha3, sizeof(double) * n * n, hipMemcpyHostToDevice);
    	hipMemcpy(db3, hb3, sizeof(double) * n * n, hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start3;
	hipEvent_t stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);

	// Lanuch the timer
	hipEventRecord(start3);

    	// Call the appropriate matmul function based on the template type
    	matmul_3(da3, db3, dc3, n, block_dim);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
    	hipDeviceSynchronize();

    	// Copy back the data stored in the device array
    	// into the host array for array c
    	hipMemcpy(hc3, dc3, sizeof(double) * n * n, hipMemcpyDeviceToHost);

	// Stop the timer
	hipEventRecord(stop3);
	hipEventSynchronize(stop3);

	// Calculate the duration time in milliseconds
    	float milliseconds3 = 0.0f;
    	hipEventElapsedTime(&milliseconds3, start3, stop3);

	// Print the first element of C
 	std::cout << hc3[0] << std::endl;

	// Print the last element of C
	std::cout << hc3[n * n - 1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
    	std::cout << milliseconds3 << std::endl;

    	// Free memory
    	delete[] ha3;
    	delete[] hb3;
    	delete[] hc3;
    	hipFree(da3);
    	hipFree(db3);
    	hipFree(dc3);

	return 0;

}




