#include "hip/hip_runtime.h"
// reduce.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "reduce.cuh"

#define threadsPerBlock 1024

// Implements the 'first add during global load' parallel reduction method
__global__ void reduce_kernel(float *g_idata, float *g_odata, unsigned int n){

    // Create shared memory for caching s_data
    __shared__ int s_data[threadsPerBlock];
    unsigned int tid = threadIdx.x;
    // Now the block size is reduced by half
    unsigned int i = threadIdx.x + blockIdx.x * (blockDim.x * 2);

    // Execute the original first iteration independently
    // The rest of the code remains unchanged
    if (i < n) {

        s_data[tid] = g_idata[i] + g_idata[i + blockDim.x];

    }

    // Synchronize threads to ensure data is loaded
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {

        if (tid < s && i + s < n) {

            s_data[tid] += s_data[tid + s];

        }

        // Synchronize threads to ensure data is loaded
        __syncthreads();
    }

    if (tid == 0) {
        g_odata[blockIdx.x] = s_data[0];
    }

}


__host__ void reduce(float **input, float **output, unsigned int N, unsigned int threads_per_block){

    // Assign the number of blocks
    unsigned int num_blocks = (N + 2 * threads_per_block - 1) / (2 * threads_per_block);

    // The first time to use reduce_kernel
    reduce_kernel<<<num_blocks, threads_per_block>>>(*input, *output, N);

    // Tell the host waits for the kernel 
    // to finish printing before returning from main
    hipDeviceSynchronize();

}


