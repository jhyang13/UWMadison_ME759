#include "hip/hip_runtime.h"
// task2.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "reduce.cuh"


int main(int argc, char **argv) {

	// Create the variables to receive the input number
	// 2^n is used to decide the size of matrix
	unsigned int N = pow(2, atoi(argv[1]));
	unsigned int threads_per_block  = atoi(argv[2]);

	// Create new array hinput
    	// Allocate memory for array hinput
	float* hinput = new float[N];

    	// Initialize the pseudo-random number generator's seed 
	// for generating random numbers in the program
    	srand(static_cast<unsigned int>(time(nullptr)));
	// Initialize array hinput with random values
	// From -1 to 1
    	for (unsigned int i = 0; i < N; ++i) {

        	hinput[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;

    	}	

    	// Create device array dinput and doutput
	float* dinput;
	float* doutput;
	// Allocate device memory for array dinput and doutput
	hipMalloc((void**)&dinput, N * sizeof(float));
    	hipMalloc((void**)&doutput, N * sizeof(float));

    	// Copy array dinput and hinput to device
    	hipMemcpy(dinput, hinput, N * sizeof(float), hipMemcpyHostToDevice);

	// Prepare CUDA timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Lanuch the timer
	hipEventRecord(start);

	// Call the reduce function
    	reduce(&dinput, &doutput, N, threads_per_block);

	// Tell the host waits for the kernel 
	// to finish printing before returning from main
    	hipDeviceSynchronize();

	// Copy the result back from device to host
    	hipMemcpy(hinput, dinput, sizeof(float), hipMemcpyDeviceToHost);

	// Stop the timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration time in milliseconds
    	float milliseconds = 0.0f;
    	hipEventElapsedTime(&milliseconds, start, stop);

	// Print the resulting sum.
 	std::cout << hinput[0] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
    	std::cout << milliseconds << std::endl;

    	// Free memory
    	delete[] hinput;
    	hipFree(dinput);
    	hipFree(doutput);

	return 0;

}




