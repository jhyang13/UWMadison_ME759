#include "hip/hip_runtime.h"
// matmul.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "matmul.cuh"

// Define the block size
#define BLOCK_SIZE 16

/*
// Without using shared memory
// Used for different types
template <typename T>
__global__ void matmul_kernel(const T* A, const T* B, T* C, unsigned int n){

    // The row index and column index of each thread in a two-dimensional grid
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Create a variable to save current result
    T sum = 0;

    for (int k = 0; k < n; k++) {

        // Create variables to save current result
        T a = A[row * n + k];
        T b = B[k * n + col];

        // Save the result
        sum += a * b;

    }

    // Assign new element to C
    C[row * n + col] = sum;

}*/


// Use shared memory
// Used for different types
template <typename T>
__global__ void matmul_kernel(const T* A, const T* B, T* C, unsigned int n){

    // The row index and column index of each thread in a two-dimensional grid
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Create shared memory for caching A and B submatrices
    __shared__ T shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ T shared_B[BLOCK_SIZE][BLOCK_SIZE];

    // Create a variable to save current result
    T sum = 0;

    // Calculate the number of submatrices
    int num_submatrices = n / BLOCK_SIZE;

    for (int i = 0; i < num_submatrices; i++) {

        // Load A and B submatrices into shared memory
        shared_A[threadIdx.y][threadIdx.x] = A[row * n + i * BLOCK_SIZE + threadIdx.x];
        shared_B[threadIdx.y][threadIdx.x] = B[(i * BLOCK_SIZE + threadIdx.y) * n + col];

        // Synchronize threads to ensure data is loaded
        __syncthreads();

        // Perform matrix multiplication on the submatrices in shared memory
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += shared_A[threadIdx.y][k] * shared_B[k][threadIdx.x];
        }

        // Synchronize threads to ensure computation is done
        __syncthreads();
    }

    // Assign the result to the appropriate element in C
    C[row * n + col] = sum;

}


__host__ void matmul_1(const int *A, const int *B, int *C, unsigned int n, unsigned int block_dim){

    // Setting the dimensions of thread blocks and grids
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);

    // Launch the matrix multiplication kernel
    matmul_kernel<<<grid, block>>>(A, B, C, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

}

__host__ void matmul_2(const float *A, const float *B, float *C, unsigned int n, unsigned int block_dim){

    // Setting the dimensions of thread blocks and grids
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);

    // Launch the matrix multiplication kernel
    matmul_kernel<<<grid, block>>>(A, B, C, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

}

__host__ void matmul_3(const double *A, const double *B, double *C, unsigned int n, unsigned int block_dim){

    // Setting the dimensions of thread blocks and grids
    dim3 block(block_dim, block_dim);
    dim3 grid((n + block.x - 1) / block.x, (n + block.y - 1) / block.y);

    // Launch the matrix multiplication kernel
    matmul_kernel<<<grid, block>>>(A, B, C, n);

    // Wait for the kernel to finish
    hipDeviceSynchronize();

}


