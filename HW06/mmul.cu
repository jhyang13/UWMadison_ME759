// mmul.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include "mmul.h"


void mmul(hipblasHandle_t handle, const float* A, const float* B, float* C, int n){

    int lda = n, ldb = n, ldc = n;
    // Initialize the scaling factor for matrices A and B
    const float Alpha = 1;
    // Initialize the scaling factor for matrix C
    const float Beta = 1;
    const float *alpha = &Alpha;
    const float *beta = &Beta;

    /*
    // Test hipblasSgemm
    hipblasStatus_t sgemmStatus = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, A, n, B, n, &beta, C, n);
    if (sgemmStatus != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS matrix multiplication failed." << std::endl;
    }*/

    // Initialize the cuBLAS context
    hipblasCreate(&handle);
    // Perform C = A * B using cuBLAS
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, alpha, A, lda, B, ldb, beta, C,  ldc);
    // Destroy the cuBLAS context
    hipblasDestroy(handle);

}




