// task1.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <hipblas.h>
#include "mmul.h"


int main(int argc, char *argv[])
{

	/*
	// Test GPU
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	if (deviceCount == 0) {
    		std::cerr << "No available GPU devices found." << std::endl;
    		return 1; // quit
	}*/

	using namespace std;
	// Create the variables to receive the input number
	int n = atol(argv[1]); 
	int n_tests = atol(argv[2]);

	// Create new matrices A, B and C
	float *A, *B, *C;
 	// Allocate memory for matrices A, B, and C
	hipMallocManaged(&A, (n*n)*sizeof(float));
	hipMallocManaged(&B, (n*n)*sizeof(float));
	hipMallocManaged(&C, (n*n)*sizeof(float));

	// Initialize matrices A and B with random values 
	// range [-1 to 1] in column-major order
	for (int i = 0; i < n*n; i++)
	{
		A[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;
		B[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;
		C[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;

	}

	// Initialize timer
	float total_time = 0.0f;

	// Calls your mmul function n tests times
	for (int run = 0; run < n_tests; run++)
	{

		// Start timing
        	hipEvent_t start, stop;
        	hipEventCreate(&start);
        	hipEventCreate(&stop);
        	hipEventRecord(start);

		// Initialize cuBLAS
		hipblasHandle_t handle;
		hipblasCreate(&handle);

		/*
		// Test
		hipblasStatus_t status = hipblasCreate(&handle);
		if (status != HIPBLAS_STATUS_SUCCESS) {
    		std::cerr << "cuBLAS initialization failed." << std::endl;
    		return 1; // quit
		}*/

		// Call mmul function
		mmul(handle, A, B, C, n); 

		// Tell the host waits for the kernel 
		// to finish printing before returning from main
		hipDeviceSynchronize();
		
		// Stop timing
        	hipEventRecord(stop);
        	hipEventSynchronize(stop);

		// Calculate the duration time in milliseconds
		float milliseconds = 0.0f;
        	hipEventElapsedTime(&milliseconds, start, stop);

		// Sum up time
		total_time += milliseconds;

	}

	// Calculate the average time
    	float average_time = total_time / n_tests;

	// Print the average time
    	std::cout << average_time << std::endl;

	// Free memory
	hipFree(A);
	hipFree(B);
	hipFree(C);

	return 0;

}
