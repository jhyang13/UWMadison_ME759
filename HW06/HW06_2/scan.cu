#include "hip/hip_runtime.h"
// scan.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "scan.cuh"

// reference: 
// https://kieber-emmons.medium.com/efficient-parallel-prefix-sum-in-metal-for-apple-m1-9e60b974d62


// FUnction to perform hillisSteeleScan
// Based on the code in lecture 14
__global__ void hillisSteeleScan(float *g_out, float *g_intput, float *g_sum, int n) 
{
    // Allocated on invocation
    extern volatile __shared__  float temp[];
    
    int thread_id = threadIdx.x;
    int index = blockIdx.x * blockDim.x + thread_id;
    int pout = 0, pin = 1;

    // Load elements to shared memory
    if (index >= n)
    { 

        // Pad with zeros if out of bounds
        temp[thread_id] = 0;

    }
    else 
    {

        // Load elements into shared memory
        temp[thread_id] = g_intput[index];
            
        // Synchronize threads
        __syncthreads();

        // Scanning within a thread group
        for (int offset = 1; offset < blockDim.x; offset *= 2)
        {

            // Swap double buffer indices
            pout = 1 - pout;
            pin = 1 - pout; 

            if ( thread_id >= offset){
                temp[pout * blockDim.x + thread_id] = temp[pin * blockDim.x + thread_id] + temp[pin * blockDim.x + thread_id - offset];
            }else{
                temp[pout * blockDim.x + thread_id] = temp[pin * blockDim.x + thread_id];
            }

            // Synchronize threads
            __syncthreads();   

        }
            
        // Store the result in output (exclusive scan)
        g_out[index] = temp[pout * n + thread_id];

         // Store the inclusive sum into the g_sum vector
        if (thread_id == blockDim.x - 1)
        { 
            g_sum[blockIdx.x] = temp[pout * n + thread_id];
        }

    }
}


// The scan function
__host__ void scan(const float* input, float* output, unsigned int n, unsigned int threads_per_block) 

{
    // Allocate cuda memory for input and ouput
    float *in; 
    float *out;

    // Allocate memory for input data
    hipMalloc(&in, n * sizeof(float));
    // Copy input data from host to device
    hipMemcpy(in, input, n * sizeof(float), hipMemcpyHostToDevice);
    // Allocate memory for output data
    hipMalloc(&out, n * sizeof(float));

    // Calculate the number of blocks
    int block_num = (n + threads_per_block - 1) / threads_per_block;
    
    float *sum;
    // Allocate CUDA memory for inclusive sum
    hipMalloc(&sum, block_num * sizeof(float));

    // Call the hillisSteeleScan function
    hillisSteeleScan<<< block_num, threads_per_block, 2 * threads_per_block * sizeof(float) >>>(out, in, sum, n);

    float *scan_sum;
    float *temp_sum;
    // Allocate CUDA memory for inclusive scan for sum
    hipMalloc(&scan_sum, block_num * sizeof(float));
    hipMalloc(&temp_sum, block_num * sizeof(float));
    
    // Copy the output from the device to the host
    hipMemcpy(output, out, n * sizeof(float), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    
    // Free the memory
    hipFree(in);
    hipFree(sum);
    hipFree(scan_sum);
    hipFree(temp_sum);
    hipFree(out);

}
