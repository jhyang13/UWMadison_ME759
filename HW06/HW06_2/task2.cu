#include "hip/hip_runtime.h"
// task2.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "scan.cuh"

int main(int argc, char *argv[]) 
{

    	// Create the variables to receive the input number
    	// 2^n is used to decide the size of matrix
	int n = pow(2, atoi(argv[1]));
	int threads_per_block  = atoi(argv[2]);

	// Create new array hinput
    	// Allocate memory for array hinput
    	float *input = new float[n];
    	float *output = new float[n];

    	// Allocate the managed memory for input and output 
    	hipMallocManaged(&input, sizeof(float) * n);
    	hipMallocManaged(&output, sizeof(float) * n);

	// Initialize the pseudo-random number generator's seed
	// for generating random numbers in the program
    	srand(static_cast<unsigned int>(time(nullptr)));
    	// Initialize array hinput with random values
	// From -1 to 1
    	for (int i = 0; i < n; i++) 
    	{
        	input[i] = static_cast<float>(rand()) / RAND_MAX * 2.0f - 1.0f;
    	}

    	// Prepare CUDA timer
    	hipEvent_t start;
    	hipEvent_t stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

    	// Start the timing
    	hipEventRecord(start);
    
	// Launch the scan kernel
	scan(input, output, n, threads_per_block);
    
	// Tell the host waits for the kernel 
	// to finish printing before returning from main
	hipDeviceSynchronize();
    
	// Stop the timing
    	hipEventRecord(stop);
    	hipEventSynchronize(stop);

	// Calculate the duration time in milliseconds
        float milliseconds = 0.0f;
        hipEventElapsedTime(&milliseconds, start, stop);

	// Print the resulting sum.
 	std::cout << output[n-1] << std::endl;

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
    	std::cout << milliseconds << std::endl;

    	// Free memory
    	hipFree(input);
    	hipFree(output);

    	return 0;

}
