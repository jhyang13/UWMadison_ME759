#include "hip/hip_runtime.h"
// task3.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <iomanip>
#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "vscale.cuh"


int main(int argc, char **argv) {

	// Create the variable to receive the input number
	unsigned int n = pow(2, atoi(argv[1]));
	
 	// Assign host memory and stack allocation for array a and b
	float ha[n];
	float hb[n];

	// Set up random seed
        srand (static_cast <unsigned> (time(0)));

	// Set decimal places before generating random numbers
	std::cout << std::fixed << std::setprecision(1);

	// Assign values to array a and b with random values
    	for (unsigned int i = 0; i < n; ++i){
        	
		// The range of values for array a is [-10.0, 10.0]
		ha[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 20.0f - 10.0f;
		
		// The range of values for array b is [0.0, 1.0]
        	hb[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX) * 1.0f;

    	}

    	// Allocate device memory for array a and b
    	float *da, *db;
    	hipMalloc((void**)&da, sizeof(float) * n);
    	hipMalloc((void**)&db, sizeof(float) * n);

    	// Copy back the data stored in the device array
        // into the host array
    	hipMemcpy(da, ha, sizeof(float) * n, hipMemcpyHostToDevice);
    	hipMemcpy(db, hb, sizeof(float) * n, hipMemcpyHostToDevice);

	// Assign 512 threads
        const int num_threads = 16;
        // Assign a 1D execution configuration that uses 512 threads per block
    	//int num_blocks = (n + num_threads - 1) / num_threads;
	int num_blocks = 1;

	// Prepare CUDA timer
	hipEvent_t start;
	hipEvent_t stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Lanuch the timer
	hipEventRecord(start);

    	// Launch the vscale kernel
    	vscale<<<num_threads, num_blocks>>>(da, db, n);

	// Tell the host waits for the kernel to finish printing before returning from main
        hipDeviceSynchronize();

    	// Copy back the data stored in the device array
        // into the host array
    	hipMemcpy(hb, db, sizeof(float) * n, hipMemcpyDeviceToHost);

	// Stop the timer
	hipEventRecord(stop);
	hipEventSynchronize(stop);

	// Calculate the duration time
	float ms = 0;
	hipEventElapsedTime(&ms, start, stop);

	// Print the amount of time taken to execute the kernel in milliseconds using CUDA event
	printf("%f\n", ms);

    	// Print the first element of the resulting array
    	std::cout << hb[0] << std::endl;

	// Print the last element of the resulting array
	std::cout << hb[n - 1] << std::endl;

    	return 0;

}





