#include "hip/hip_runtime.h"
// task3 
// vscale.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <cstdio>
#include <ctime>
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "vscale.cuh"


// Function that does an element-wise multiplication 
// of the two arrays: bi = ai · bi
__global__ void vscale(const float *a, float *b, unsigned int n){

	// Assign CUDA and the global unique index of a thread within the grid
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	// Do an element-wise multiplication of the two arrays
	for (int i = index; i < n; i += blockDim.x * gridDim.x) {
        	
		b[i] = a[i] * b[i];

   	}	
}

