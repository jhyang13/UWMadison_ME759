// task2.cu

/*After completing my test,
I requested suggestions from ChatGPT
for improving the code.*/

#include <cstdio>
#include <iostream>
#include <string>
#include <random>
#include <hip/hip_runtime.h>

// Defined GPU Kernel Function
__global__ void simpleKernel(int* dA, int a) {

	// Assign Cuda and the global unique index of a thread within the grid
    	int x = threadIdx.x;
    	int y = blockIdx.x;
    	
	// Create the array dA with sixteen ints on the device
	// Compute ax+y on each thread 
	// Write the result in one distinct entry of the dA array	
	dA[y * blockDim.x + x] = a * x + y;

}


int main() {

	// Assign eight threads
    	const int num_threads = 8;
	// Assign two blocks
    	const int num_blocks = 2;

	// Define the array size for the host array
    	const int asize = num_threads * num_blocks;
    
        // Generate the random integer a
        const int RANGE = 50;
        int a = rand() % (RANGE + 1);

    	// Create a new array hA
    	int hA[asize];
	// Create a pointer to dA
    	int* dA;
    
    	// Allocate memory space on the device for storing dA
    	hipMalloc((void**)&dA, sizeof(int) * asize);
    
    	// Invoke GPU kernel, with two block and eight threads
    	simpleKernel<<<num_blocks, num_threads>>>(dA, a);
    
    	// Copy back the data stored in the device array dA 
        // into the host array hA
    	hipMemcpy(hA, dA, sizeof(int) * asize, hipMemcpyDeviceToHost);
    
    	// Loop to print (from the host) the sixteen values 
	// stored in the host array separated by a single space each
    	for (int i = 0; i < asize; ++i) {

        	std::cout << hA[i] << " ";

    	}
    
    	// Deallocate the memory on the device 
	// that was previously allocated using cudaMalloc
    	hipFree(dA);
    
    	return 0;

}

