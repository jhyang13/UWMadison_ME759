// task1.cu

/*After completing my test, 
I requested suggestions from ChatGPT 
for improving the code.*/

#include <cstdio>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

// Defined GPU Kernel Function
__global__ void simpleKernel() {

	// Assign CUDA and the global unique index of a thread within the grid
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	// Assign value to a and b
	int a = index + 1;
	int b = 1;

    	// Calculate the factorial b = a!
    	for (int i = 1; i < a+1; i++) {

        	b = b * i;
    	}	

    	// Print out the result
	printf("%d!=%d\n", a, b);
}



int main() {

	// Assign eight threads
    	const int num_threads = 8;
	// Assign one block
	const int num_blocks = 1;

	// Invoke GPU kernel, with one block and eight threads
    	simpleKernel<<<num_blocks, num_threads>>>();

	// Tell the host waits for the kernel to finish printing before returning from main
    	hipDeviceSynchronize();

    	return 0;
}




